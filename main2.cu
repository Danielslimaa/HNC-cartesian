#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include "kernels.cuh"

int main(void)
{
  h_N = 1 << 7;
  double h_L = 10;
  double h_h = h_L / h_N;
  double h_rho = 1;
  double h_dx = h_L / h_N;
  double h_dy = h_dx;
  double h_dk = M_PI / h_L;
  double h_dkx = h_dk;
  double h_dky = h_dkx;
  double h_dt;
  double U = 10.;

  h_dt = 0.01;
  hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(int), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(h), &h_h, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(L), &h_L, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rho), &h_rho, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dx), &h_dx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dy), &h_dy, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dkx), &h_dkx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dky), &h_dky, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_dt, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  printf("N = %d, h = %f, L = %f\n", h_N, h_h, h_L);
  printer_constants<<<1,1>>>();

  int Blocks_N, ThreadsPerBlock_N;
  if (h_N * h_N >= 1024)
  {
    ThreadsPerBlock_N = 1024;
  }
  else
  {
    ThreadsPerBlock_N = h_N * h_N;
  }
  Blocks_N = (int)ceil((double)(h_N * h_N) / 1024.0);

  printf("Blocks_N = %d, ThreadsPerBlock_N = %d\n", Blocks_N, ThreadsPerBlock_N);

  double * x, * y, * kx, * ky, * k2, * V, * g, * S, *new_S, * omega, * Vph, * second_term;
  hipMalloc(&x, sizeof(double) * h_N * h_N);
  hipMalloc(&y, sizeof(double) * h_N * h_N);
  hipMalloc(&kx, sizeof(double) * h_N * h_N);
  hipMalloc(&ky, sizeof(double) * h_N * h_N);
  hipMalloc(&k2, sizeof(double) * h_N * h_N);
  hipMalloc(&V, sizeof(double) * h_N * h_N);
  hipMalloc(&g, sizeof(double) * h_N * h_N);
  hipMalloc(&S, sizeof(double) * h_N * h_N);
  hipMalloc(&new_S, sizeof(double) * h_N * h_N);
  hipMalloc(&omega, sizeof(double) * h_N * h_N);
  hipMalloc(&Vph, sizeof(double) * h_N * h_N);
  hipMalloc(&second_term, sizeof(double) * h_N * h_N);

  double * h_x = new double[h_N * h_N];
  double * h_y = new double[h_N * h_N];
  double * h_kx = new double[h_N * h_N];
  double * h_ky = new double[h_N * h_N];
  double * h_k2 = new double[h_N * h_N];

  #pragma omp parallel for
  for (int i = 0; i < h_N; i++)
  {
    for (int j = 0; j < h_N; j++)
    {
      h_x[i * h_N + j] = (0) + i * h_dx;
      h_y[i * h_N + j] = (0) + j * h_dy;
      h_kx[i * h_N + j] = (0) + i * h_dkx;
      h_ky[i * h_N + j] = (0) + j * h_dky;
      h_k2[i * h_N + j] = h_kx[i * h_N + j] * h_kx[i * h_N + j] + h_ky[i * h_N + j] * h_ky[i * h_N + j];
    }
  }
  hipMemcpy(x, h_x, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(y, h_y, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(kx, h_kx, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(ky, h_ky, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(k2, h_k2, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  double * h_V = new double[h_N * h_N];
  double * tmp = new double[h_N * h_N];
  #pragma omp parallel for
  for (int i = 0; i < h_N * h_N; i++)
  {
    h_V[i] = U * exp( -h_x[i] * h_x[i] - h_y[i] * h_y[i] );
    tmp[i] = 1;//exp( -h_x[i] * h_x[i] - h_y[i] * h_y[i] );
  }  
  hipMemcpy(V, h_V, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  printer_vector(h_x, h_y, V, "U.dat", h_N);  

  hipMemcpy(g, tmp, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(S, tmp, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  delete[] tmp;
  printer_vector(h_x, h_y, g, "g0.dat", h_N);
  int * h_index = new int[h_N];
  for (int i = 0; i < h_N; ++i) 
  {
    h_index[i] = i;
  }

  int * index;
  hipMalloc(&index, sizeof(int) * h_N);
  hipMemcpy(index, h_index, sizeof(int) * h_N, hipMemcpyHostToDevice);
  delete[] h_index;

  /*
  //FFT_x<<<Blocks_N, ThreadsPerBlock_N>>>(g, x, kx);
  hipDeviceSynchronize();
  FFT_y<<<Blocks_N, ThreadsPerBlock_N>>>(g, y, ky);
  hipDeviceSynchronize();
  //laplace<<<Blocks_N, ThreadsPerBlock_N>>>(k2,g);
  hipDeviceSynchronize();
  //IFFT_x<<<Blocks_N, ThreadsPerBlock_N>>>(g, x, kx);
  hipDeviceSynchronize();
  IFFT_y<<<Blocks_N, ThreadsPerBlock_N>>>(g, y, ky);  
  hipDeviceSynchronize();
  */

  ffty_test<<<1, h_N>>>(S, g, &index[5]);
  printer_array(S, "S1.dat", h_N);
  iffty_test<<<1, h_N>>>(S, g, &index[5]);
  
  //IFFT_y<<<1, h_N>>>(g, y, ky, &index[2]);

  printer_array(g, "g.dat", h_N);
  printer_array(S, "S2.dat", h_N);
  //printer_vector(h_x, h_y, g, "g.dat", h_N);

  delete[] h_x;
  delete[] h_y;
  delete[] h_V;
  delete[] h_k2;
  hipFree(x);
  hipFree(y);
  hipFree(kx);
  hipFree(ky);
  hipFree(k2);
  hipFree(V);
  hipFree(g);
  hipFree(S);
  hipFree(new_S);
  hipFree(omega);
  hipFree(Vph);
  hipFree(second_term);
  hipFree(index);
  hipDeviceReset();
  return 0;
}