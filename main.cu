#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include "kernels.cuh"

int main(void){
  h_N = 1 << 8;
  double h_L = 5;
  double h_h = h_L / h_N;
  double h_rho = 1;
  double h_dx = h_L / h_N;
  double h_dy = h_dx;
  double h_dk = 13.0 / h_N;
  double h_dkx = h_dk;
  double h_dky = h_dkx;
  hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(int), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(h), &h_h, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(L), &h_L, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rho), &h_rho, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dx), &h_dx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dy), &h_dy, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dkx), &h_dkx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL("dky"), &h_dky, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  printf("N = %d, h = %f, L = %f\n", h_N, h_h, h_L);

  int Blocks_N, ThreadsPerBlock_N;
  if (h_N * h_N >= 1024)
  {
    ThreadsPerBlock_N = 1024;
  }
  else
  {
    ThreadsPerBlock_N = h_N * h_N;
  }
  Blocks_N = (int)ceil((double)(h_N * h_N) / 1024.0);

  printf("Blocks_N = %d, ThreadsPerBlock_N = %d\n", Blocks_N, ThreadsPerBlock_N);

  dim3 threadsPerBlock(h_N, h_N);
  dim3 numBlocks(h_N / threadsPerBlock.x, h_N / threadsPerBlock.y);

  printf("numBlocks = (%d, %d)\n", h_N / threadsPerBlock.x, h_N / threadsPerBlock.y);

  double * U, * g, * S;
  hipMalloc(&U, sizeof(double) * h_N * h_N);
  hipMalloc(&S, sizeof(double) * h_N * h_N);
  hipMalloc(&g, sizeof(double) * h_N * h_N);

  double * x = new double[h_N * h_N];
  double * y = new double[h_N * h_N];
  double * kx = new double[h_N * h_N];
  double * ky = new double[h_N * h_N];

  #pragma omp parallel for
  for (int i = 0; i < h_N; i++)
  {
    for (int j = 0; j < h_N; j++)
    {
      x[i * h_N + j] = (0) + (i - 1) * h_dx;
      y[i * h_N + j] = (0) + (j - 1) * h_dy;
      kx[i * h_N + j] = (0) + (i - 1) * h_dkx;
      ky[i * h_N + j] = (0) + (j - 1) * h_dky;
    }
  }

  double * h_U = new double[h_N * h_N];
  #pragma omp parallel for
  for (int i = 0; i < h_N * h_N; i++)
  {
    h_U[i] = exp( -x[i] * x[i] - y[i] * y[i] );
  }  
  hipMemcpy(U, h_U, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  printer_vector(x, y, U, "U.dat", h_N);  

  hipMemcpy(g, h_U, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(S, h_U, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);

  printer_vector(x, y, g, "g0.dat", h_N);

  numStreams = h_N; // Number of CUDA streams
  // Allocate memory for the array of streams
  hipStream_t * streams_y = new hipStream_t[numStreams];
  hipStream_t * streams_x = new hipStream_t[numStreams];
  // Create each stream
  printf("Create each stream\n");
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamCreate(&streams_y[i]));
    CUDA_CHECK(hipStreamCreate(&streams_x[i]));
  }  
  printf("Streams created.\n");
  
  FFT(g, S, streams_x, streams_y, numBlocks, threadsPerBlock);


  printer_vector(x, y, g, "g2.dat", h_N);
  // Destroy each stream
  printf("Destroy each stream\n");
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamDestroy(streams_y[i]));
    CUDA_CHECK(hipStreamDestroy(streams_x[i]));
  }
  
  delete[] x;
  delete[] y;
  delete[] h_U;
  hipFree(g);
  hipFree(S);
  hipFree(index);
  delete[] streams_y;
  delete[] streams_x;
  hipDeviceReset();
  return 0;
}