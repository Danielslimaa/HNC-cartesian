#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include "kernels.cuh"

int main(void)
{
  h_N = 1 << 8;
  double h_L = 10;
  double h_h = h_L / h_N;
  double h_rho = 1;
  double h_dx = h_L / h_N;
  double h_dy = h_dx;
  double h_dk = 5.0 / h_N;
  double h_dkx = h_dk;
  double h_dky = h_dkx;
  double h_dt;
  double U = 10.;

  h_dt = 0.001;
  hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(int), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(h), &h_h, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(L), &h_L, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rho), &h_rho, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dx), &h_dx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dy), &h_dy, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dkx), &h_dkx, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dky), &h_dky, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_dt, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  printf("N = %d, h = %f, L = %f\n", h_N, h_h, h_L);

  int Blocks_N, ThreadsPerBlock_N;
  if (h_N * h_N >= 1024)
  {
    ThreadsPerBlock_N = 1024;
  }
  else
  {
    ThreadsPerBlock_N = h_N * h_N;
  }
  Blocks_N = (int)ceil((double)(h_N * h_N) / 1024.0);

  printf("Blocks_N = %d, ThreadsPerBlock_N = %d\n", Blocks_N, ThreadsPerBlock_N);

  dim3 threadsPerBlock(h_N, h_N);
  dim3 numBlocks(h_N / threadsPerBlock.x, h_N / threadsPerBlock.y);

  printf("numBlocks = (%d, %d)\n", h_N / threadsPerBlock.x, h_N / threadsPerBlock.y);

  double * k2, * V, * g, * S, *new_S, * omega, * Vph, * second_term;
  hipMalloc(&k2, sizeof(double) * h_N * h_N);
  hipMalloc(&V, sizeof(double) * h_N * h_N);
  hipMalloc(&g, sizeof(double) * h_N * h_N);
  hipMalloc(&S, sizeof(double) * h_N * h_N);
  hipMalloc(&new_S, sizeof(double) * h_N * h_N);
  hipMalloc(&omega, sizeof(double) * h_N * h_N);
  hipMalloc(&Vph, sizeof(double) * h_N * h_N);
  hipMalloc(&second_term, sizeof(double) * h_N * h_N);

  double * x = new double[h_N * h_N];
  double * y = new double[h_N * h_N];
  double * kx = new double[h_N * h_N];
  double * ky = new double[h_N * h_N];
  double * h_k2 = new double[h_N * h_N];

  #pragma omp parallel for
  for (int i = 0; i < h_N; i++)
  {
    for (int j = 0; j < h_N; j++)
    {
      x[i * h_N + j] = (0) + (i - 1) * h_dx;
      y[i * h_N + j] = (0) + (j - 1) * h_dy;
      kx[i * h_N + j] = (0) + (i - 1) * h_dkx;
      ky[i * h_N + j] = (0) + (j - 1) * h_dky;
      h_k2[i * h_N + j] = kx[i * h_N + j] * kx[i * h_N + j] + ky[i * h_N + j] * ky[i * h_N + j];
    }
  }
  hipMemcpy(k2, h_k2, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  double * h_V = new double[h_N * h_N];
  double * tmp = new double[h_N * h_N];
  #pragma omp parallel for
  for (int i = 0; i < h_N * h_N; i++)
  {
    h_V[i] = U * exp( -x[i] * x[i] - y[i] * y[i] );
    tmp[i] = 1.0;
  }  
  hipMemcpy(V, h_V, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  printer_vector(x, y, V, "U.dat", h_N);  


  hipMemcpy(g, tmp, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  hipMemcpy(S, tmp, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  delete[] tmp;
  printer_vector(x, y, g, "g0.dat", h_N);

  numStreams = h_N; // Number of CUDA streams
  // Allocate memory for the array of streams
  hipStream_t * streams_y = new hipStream_t[numStreams];
  hipStream_t * streams_x = new hipStream_t[numStreams];
  hipEvent_t * events_x = new hipEvent_t[numStreams];
  hipEvent_t * events_y = new hipEvent_t[numStreams];
  // Create each stream
  printf("Creating streams and cudaEvents\n");
  int * h_index = new int[h_N];
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamCreate(&streams_y[i]));
    CUDA_CHECK(hipStreamCreate(&streams_x[i]));
    CUDA_CHECK(hipEventCreate(&events_x[i]));
    CUDA_CHECK(hipEventCreate(&events_y[i]));
    h_index[i] = i;
  }  
  printf("Streams created.\n");
  int * index;
  hipMalloc(&index, sizeof(int) * h_N);
  hipMemcpy(index, h_index, sizeof(int) * h_N, hipMemcpyHostToDevice);
  delete[] h_index;
  bool condition = true;
  long int counter = 1;
  while(counter < 4)
  {    
    compute_second_term(g, second_term, numBlocks, threadsPerBlock);
    compute_omega(omega, k2, g, S, events_x, events_y, streams_x, streams_y, numBlocks, threadsPerBlock, index);
    compute_Vph_k(V, second_term, g, omega, Vph, events_x, events_y, streams_x, streams_y, numBlocks, threadsPerBlock, index);
    update_S<<<Blocks_N, ThreadsPerBlock_N>>>(S, k2, Vph);
    IFFT_S2g(g, S, events_x, events_y, streams_x, streams_y, numBlocks, threadsPerBlock, index);
    printf("counter = %ld\n", counter);
    counter++;
  }  

  printer_vector(x, y, g, "g.dat", h_N);
  // Destroy each stream
  printf("Destroy each stream\n");
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamDestroy(streams_y[i]));
    CUDA_CHECK(hipStreamDestroy(streams_x[i]));
    CUDA_CHECK(hipEventDestroy(events_x[i]));
    CUDA_CHECK(hipEventDestroy(events_y[i]));
  }
  delete[] streams_y;
  delete[] streams_x;
  
  delete[] x;
  delete[] y;
  delete[] h_V;
  delete[] h_k2;
  hipFree(k2);
  hipFree(V);
  hipFree(g);
  hipFree(S);
  hipFree(new_S);
  hipFree(omega);
  hipFree(Vph);
  hipFree(second_term);
  hipFree(index);
  hipDeviceReset();
  return 0;
}