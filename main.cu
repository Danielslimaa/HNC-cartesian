#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <cstdlib>
#include <omp.h>
#include "kernels.cuh"

int main(void){
  int h_N = 1 << 8;
  double h_L = 40;
  double h_h = h_L / h_N;
  double h_rho = 1;
  hipMemcpyToSymbol(HIP_SYMBOL(N), &h_N, sizeof(int), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(h), &h_h, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(L), &h_L, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rho), &h_rho, sizeof(double), size_t(0), hipMemcpyHostToDevice);
  printf("N = %d, h = %f, L = %f\n", h_N, h_h, h_L);

  int Blocks_N, ThreadsPerBlock_N;
  if (h_N * h_N >= 1024)
  {
    ThreadsPerBlock_N = 1024;
  }
  else
  {
    ThreadsPerBlock_N = h_N * h_N;
  }
  Blocks_N = (int)ceil((double)(h_N * h_N) / 1024.0);

  printf("Blocks_N = %d, ThreadsPerBlock_N = %d\n", Blocks_N, ThreadsPerBlock_N);

  double * U;
  hipMalloc(&U, sizeof(double) * h_N * h_N);

  double * x = new double[h_N * h_N];
  double * y = new double[h_N * h_N];

  #pragma omp parallel for
  for (int i = 0; i < h_N; i++)
  {
    for (int j = 0; j < h_N; j++)
    {
      x[i * h_N + j] = (-h_L / 2.) + (i - 1) * h_h;
      y[i * h_N + j] = (-h_L / 2.) + (j - 1) * h_h;
    }
  }

  double * h_U = new double[h_N * h_N];
  #pragma omp parallel for
  for (int i = 0; i < h_N * h_N; i++)
  {
    h_U[i] = exp( -x[i] * x[i] - y[i] * y[i] );
  }  
  hipMemcpy(U, h_U, sizeof(double) * h_N * h_N, hipMemcpyHostToDevice);
  printer_vector(x, y, U, "U.dat", h_N);  

  const int numStreams = N; // Number of CUDA streams
  // Allocate memory for the array of streams
  hipStream_t* streams_y = new hipStream_t[numStreams];
  hipStream_t* streams_x = new hipStream_t[numStreams];

  // Create each stream
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamCreate(&streams_y[i]));
    CUDA_CHECK(hipStreamCreate(&streams_x[i]));
  }  


  dim3 threadsPerBlock(h_N, h_N);
  dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
  DCT_x<<<numBlocks, threadsPerBlock>>>(U, U);
  DCT_y<<<numBlocks, threadsPerBlock>>>(U, U);
  printer_vector(x, y, U, "FFT_U.dat", h_N);  
  DCT_x<<<numBlocks, threadsPerBlock>>>(U, U);
  DCT_y<<<numBlocks, threadsPerBlock>>>(U, U);
  rescaling<<<Blocks_N, ThreadsPerBlock_N>>>(U);
  printer_vector(x, y, U, "IFFT_FFT_U.dat", h_N);                              

  // Synchronize and destroy each stream
  for (int i = 0; i < numStreams; ++i) 
  {
    CUDA_CHECK(hipStreamDestroy(streams_y[i]));
    CUDA_CHECK(hipStreamDestroy(streams_x[i]));
  }

  hipDeviceReset();
  delete[] x;
  delete[] y;
  delete[] h_U;
  delete[] streams_y;
  delete[] streams_x;
  return 0;
}